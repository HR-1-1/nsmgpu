/************************************************************
* Author   : Harish R
* Created  : June 26 2022
* Modified : June 26 2022
* Purpose  : Memory sharing between CPU & GPU
*************************************************************/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100

__global__ void dkernel(int a[]){
    a[threadIdx.x] = threadIdx.x*threadIdx.x;
}
int main(){
    int a[N], *da;

    hipMalloc(&da, N*sizeof(int));
    dkernel<<<1,N>>>(da);
    hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i=0; i<N; i++){
        printf("%d\n", a[i]);
    }
    hipDeviceSynchronize();
    return 0;
}
